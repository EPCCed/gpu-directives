
#include <hip/hip_runtime.h>
#include <iostream>

using real_t = double;

// blockDim.x - num threads in a block, .x indicates 1D block labelling
// blockIdx.x - thread index number
// multiplying the above two variables gives start of block
// then add the threadIdx.x offset for the particular thread

__global__ void saxpy_parallel(int n, real_t a, real_t *x, real_t *y)
{


	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i<n)  y[i] = a*x[i] + y[i];


}

void checkCUDAError(const char *msg);

int main()
{
	int N =10;
    int nTrials = 100;
    double tol = 1e-5;


	// allocate vectors on host
	size_t data_size = N * sizeof(real_t);
	real_t* h_x = new real_t[data_size];
	real_t* h_y = new real_t[data_size];

	// allocate device memory
	real_t* d_x; real_t* d_y;

	hipMalloc( &d_x, data_size);
	hipMalloc( &d_y, data_size);

    
	for (int i = 0;i<=N-1;i++)
	{
        h_x[i]=1;
        h_y[i]=i;
        
		//std::cout << i << " " <<  h_y[i] << std::endl;
	}

    hipMemcpy(d_x, h_x, data_size, hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, data_size, hipMemcpyHostToDevice);	

	// calculate number of blocks needed for N 
	int nblocks = (N+255)/256;


    std::cout << "Start calculation" << std::endl;

    for(int iTrial=0;iTrial<nTrials;iTrial++)
    {
        saxpy_parallel<<<nblocks,256>>>(N,2.0,d_x,d_y);
    }
    std::cout << "End calculation" << std::endl;
    checkCUDAError("kernel execution calls");


// 	// Copy results back from device memory to host memory
// 	// implicty waits for threads to excute
 	hipMemcpy(h_y, d_y, data_size, hipMemcpyDeviceToHost);

	// Check for any CUDA errors
    checkCUDAError("hipMemcpy calls");

	for (int i = 0;i<=N-1;i++)
	{
        real_t expected = i + nTrials*2;

		if (std::abs( h_y[i] - expected ) > tol )
        {
            std::cout << "Error at " << i << ". Expected "<< expected << " but got " << h_y[i]<<std::endl;
            exit(1);
        }
	}



//   cudaFree(d_x);
//   cudaFree(d_y);

//   delete h_x;
//   delete h_y;

    std::cout << "Completed"<< std::endl;

  return 0;

}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }                         
}